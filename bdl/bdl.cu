// nvcc -arch=native -O3 bdl.cu -o bdl
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>

#define CUDA_CALL( ... ) do {                                               \
    __VA_ARGS__;                                                            \
    hipError_t __res = hipGetLastError();                                 \
    if (hipSuccess != __res)                                               \
        fprintf(stderr, "CUDA error %i in %s:%d: %s (%s)\n",                \
        __res, __FILE__, __LINE__, hipGetErrorString(__res), #__VA_ARGS__);\
} while(0)

using ll = long long;
using ull = unsigned long long;

struct poly { ull x1, x0; };

ull base = 147100243658956343ull;
ull _mod = 7759751235135804287ull;
ull _mod2 = 4189298803ull;
__device__ ull mod;
__device__ ull mod2;
__device__ ull _x;
poly *__poly;

int sz;
constexpr int mx = INT_MAX;

__global__ void device_ctors(ull x)
{
    mod = 7759751235135804287ull;
    mod2 = 4189298803ull;
    _x = x;
}

__global__ void kernel(poly* _poly, int start)
{
	start += threadIdx.x;
	ull in = start;
    ull s2 = 0;

    in *= 123456789;
    in %= mod2;

    s2 ^= ((in*_poly[0].x1)%mod2+_poly[0].x0)%mod2;
    s2 ^= ((in*_poly[1].x1)%mod2+_poly[1].x0)%mod2;
    s2 ^= ((in*_poly[2].x1)%mod2+_poly[2].x0)%mod2;
    s2 ^= ((in*_poly[3].x1)%mod2+_poly[3].x0)%mod2;
    s2 ^= ((in*_poly[4].x1)%mod2+_poly[4].x0)%mod2;
    s2 ^= ((in*_poly[5].x1)%mod2+_poly[5].x0)%mod2;
    s2 ^= ((in*_poly[6].x1)%mod2+_poly[6].x0)%mod2;
    s2 ^= ((in*_poly[7].x1)%mod2+_poly[7].x0)%mod2;
    s2 ^= ((in*_poly[8].x1)%mod2+_poly[8].x0)%mod2;
    s2 ^= ((in*_poly[9].x1)%mod2+_poly[9].x0)%mod2;
    in = (in*_poly[9].x1+_poly[9].x0)%mod2;
    in = s2;
    s2 ^= ((in*_poly[0].x1)%mod2+_poly[0].x0)%mod2;
    s2 ^= ((in*_poly[1].x1)%mod2+_poly[1].x0)%mod2;
    s2 ^= ((in*_poly[2].x1)%mod2+_poly[2].x0)%mod2;
    s2 ^= ((in*_poly[3].x1)%mod2+_poly[3].x0)%mod2;
    s2 ^= ((in*_poly[4].x1)%mod2+_poly[4].x0)%mod2;
    s2 ^= ((in*_poly[5].x1)%mod2+_poly[5].x0)%mod2;
    s2 ^= ((in*_poly[6].x1)%mod2+_poly[6].x0)%mod2;
    s2 ^= ((in*_poly[7].x1)%mod2+_poly[7].x0)%mod2;
    s2 ^= ((in*_poly[8].x1)%mod2+_poly[8].x0)%mod2;
    s2 ^= ((in*_poly[9].x1)%mod2+_poly[9].x0)%mod2;
    in = (in*_poly[9].x1+_poly[9].x0)%mod2;
    s2 *= s2;

    if (s2 == _x) {
        printf("Found %d\n", start);
    }
}

inline ull fpow(ull pod, ull wyk)
{
    ull w = 1;
    wyk %= _mod;
    while (wyk)
    {
        if (wyk & 1)
        {
            w *= pod;
            w %= _mod;
        }
        pod *= pod;
        pod %= _mod;
        wyk /= 2;
    }
    return w;
}

inline ull c(ull b, int z)
{
    return __builtin_popcountll(fpow(b >> (z+1), b & ((1ull << z) - 1)) >> 1)%2;
}

int main()
{
	hipDeviceProp_t properties;
	CUDA_CALL( hipGetDeviceProperties(&properties, 0) );
	sz = properties.maxThreadsPerMultiProcessor;
	
	printf("::: Calculating f^-1(x) on GPU0 (%s) using %d cores.\n", properties.name, sz);
	printf("::: x = %llu, mod = %llu, mod2 = %llu\n", base, _mod, _mod2);
    printf("Memoazing rand()...\n");

	srand(_mod2);
    ll offset = 0;
    for (int i = 0; i < 100; i++)
        offset += (ull)rand() * (ull)(rand() % 2 ? 1 : -1);

    printf("rand() result: %lld\n", offset);
    printf("Multiplying xor-shift polynomials...\n");

    poly p {1, 0};
    poly coefs[10];
    int coefs_i = 0;
    for (ull i=2; i<100000001; i++) {
        if (i % 2 == 1) {
            p.x0 *= i;
            p.x0 %= _mod2;
            p.x1 *= i;
            p.x1 %= _mod2;
        } else {
            p.x0 += i;
            p.x0 %= _mod2;
        }
        if (i % 10000000 == 0) {
            printf("P_%d = (%-10llu x^1, %-10llu x^0)\n", coefs_i, p.x1, p.x0);
            coefs[coefs_i++] = p;
        }
    }
    coefs[coefs_i++] = p;

    printf("Uploading to GPU...\n");

	CUDA_CALL( hipMalloc(&__poly, 10*sizeof(*coefs)) );
	CUDA_CALL( hipMemcpy(__poly, coefs, 10*sizeof(*coefs), hipMemcpyHostToDevice) );

    printf("Reversing the power-log function...\n");

    ull x = base - offset;

	for (int k = 0; k < 654321; k++)
    {
        for (int i=62; i>=0; i--)
        {
            ull a = x ^ (1ull << i);
            ull b = x;
            if (x == (b ^ ((1ull << i) * c(b, i)))) {
                x = b;
            } else if (x == (a ^ ((1ull << i) * c(a, i)))) {
                x = a;
            } else {
                printf("UNREACHABLE!");
                exit(-1);
            }
        }
    }

    printf("transformed %llu -> %llu\n", base-offset, x);
    CUDA_CALL( device_ctors<<<1, 1>>>(x) );

    printf("Running search from 0 to %d, using 1 grid(s) %d threads\n", mx, sz);
	for (ull i=0; i<mx; i+=sz) {
		CUDA_CALL( kernel<<<1, sz>>>(__poly, i) );
	}
	hipDeviceSynchronize();
}